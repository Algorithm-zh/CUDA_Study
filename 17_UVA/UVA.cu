#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/freshman.h"


void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);

  int nElem=1<<14;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *res_from_gpu_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_host,*b_host,*res_d;
  //统一虚拟寻址
  //设备架构2.0后，有了UVA（同一寻址方式）的内存机制，设备内存和主机内存被映射到同一虚拟内存地址空间
  //这样搞之后就不需要用hipHostGetDevicePointer这个函数专门再取获得一个设备变量来用了
  //可以直接将分配的固定主机内存地址传递给核函数
  CHECK(hipHostAlloc((float**)&a_host,nByte,hipHostMallocMapped));
  CHECK(hipHostAlloc((float**)&b_host,nByte,hipHostMallocMapped));
  CHECK(hipMalloc((float**)&res_d,nByte));
  res_from_gpu_h=(float*)malloc(nByte);

  initialData(a_host,nElem);
  initialData(b_host,nElem);

  dim3 block(1024);
  dim3 grid(nElem/block.x);
  sumArraysGPU<<<grid,block>>>(a_host,b_host,res_d);
  printf("Execution configuration<<<%d,%d>>>\n",grid.x,block.x);

  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  sumArrays(a_host,b_host,res_h,nElem);

  checkResult(res_h,res_from_gpu_h,nElem);
  hipHostFree(a_host);
  hipHostFree(b_host);
  hipFree(res_d);

  free(res_h);
  free(res_from_gpu_h);

  return 0;
}
