#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/freshman.h"
//cpu transform
void transformMatrix2D_CPU(float * MatA,float * MatB,int nx,int ny)
{
  for(int j=0;j<ny;j++)
  {
    for(int i=0;i<nx;i++)
    {
      MatB[i*nx+j]=MatA[j*nx+i];
    }
  }
}
//按行读(比按列快的多，因为合并读取)
__global__ void copyRow(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*nx;
    if (ix<nx && iy<ny)
    {
      MatB[idx]=MatA[idx];
    }
}
//按列读
__global__ void copyCol(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx]=MatA[idx];
    }
}
//按行朴素转置
__global__ void transformNaiveRow(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx_col]=MatA[idx_row];
    }
}
//按列朴素转置(按列比按行转置快，因为L1会缓存，缓存了的数据就不用在读取)
__global__ void transformNaiveCol(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx_row]=MatA[idx_col];
    }
}
//按行展开转置(展开后发现无论行还是列都会比最开始的按行读更快)
__global__ void transformNaiveRowUnroll(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x*4;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx_col]=MatA[idx_row];
      //对 B 来说，行号 = ix，往右走 blockDim.x，就是「行号增加了 blockDim.x」：
      MatB[idx_col+ny*1*blockDim.x]=MatA[idx_row+1*blockDim.x];//B[(ix+blockDim.x)∗ny+iy]
      MatB[idx_col+ny*2*blockDim.x]=MatA[idx_row+2*blockDim.x];
      MatB[idx_col+ny*3*blockDim.x]=MatA[idx_row+3*blockDim.x];
    }
}
//按列展开转置
__global__ void transformNaiveColUnroll(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x*4;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
        MatB[idx_row]=MatA[idx_col];
        MatB[idx_row+1*blockDim.x]=MatA[idx_col+ny*1*blockDim.x];
        MatB[idx_row+2*blockDim.x]=MatA[idx_col+ny*2*blockDim.x];
        MatB[idx_row+3*blockDim.x]=MatA[idx_col+ny*3*blockDim.x];
    }
}
//对角读取，使用 f(x,y)=(m,n)函数来将坐标打乱，是为了打乱线程块，防止连续的线程块访问相近的DRAM，
//因为DRAM过多的访问同一个分区(256个字节一个分区)的话可能会出现排队的情况，所以让他分散开,.调整块的ID
__global__ void transformNaiveRowDiagonal(float * MatA,float * MatB,int nx,int ny)
{
    int block_y=blockIdx.x;
    int block_x=(blockIdx.x+blockIdx.y)%gridDim.x;
    int ix=threadIdx.x+blockDim.x*block_x;
    int iy=threadIdx.y+blockDim.y*block_y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx_col]=MatA[idx_row];
    }
}
__global__ void transformNaiveColDiagonal(float * MatA,float * MatB,int nx,int ny)
{
    int block_y=blockIdx.x;
    int block_x=(blockIdx.x+blockIdx.y)%gridDim.x;
    int ix=threadIdx.x+blockDim.x*block_x;
    int iy=threadIdx.y+blockDim.y*block_y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      MatB[idx_row]=MatA[idx_col];
    }
}



int main(int argc,char** argv)
{
  printf("strating...\n");
  initDevice(0);
  int nx=1<<12;
  int ny=1<<12;
  int dimx=32;
  int dimy=32;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);
  int transform_kernel=0;
  if(argc==2)
    transform_kernel=atoi(argv[1]);
  if(argc>=4)
  {
      transform_kernel=atoi(argv[1]);
      dimx=atoi(argv[2]);
      dimy=atoi(argv[3]);
  }

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  float* B_host=(float*)malloc(nBytes);
  initialData(A_host,nxy);

  //hipMalloc
  float *A_dev=NULL;
  float *B_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));
  CHECK(hipMalloc((void**)&B_dev,nBytes));

  CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  CHECK(hipMemset(B_dev,0,nBytes));



  // cpu compute
  double iStart=cpuSecond();
  transformMatrix2D_CPU(A_host,B_host,nx,ny);
  double iElaps=cpuSecond()-iStart;
  printf("CPU Execution Time elapsed %f sec\n",iElaps);

  // 2d block and 2d grid
  dim3 block(dimx,dimy);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);
  dim3 block_1(dimx,dimy);
  dim3 grid_1((nx-1)/(block_1.x*4)+1,(ny-1)/block_1.y+1);
  iStart=cpuSecond();
  switch(transform_kernel)
  {
  case 0:
    copyRow<<<grid,block>>>(A_dev,B_dev,nx,ny);
    break;
  case 1:
    copyCol<<<grid,block>>>(A_dev,B_dev,nx,ny);
    break;
  case 2:
    transformNaiveRow<<<grid,block>>>(A_dev,B_dev,nx,ny);
    break;
  case 3:
        transformNaiveCol<<<grid,block>>>(A_dev,B_dev,nx,ny);
        break;
  case 4:
        transformNaiveColUnroll<<<grid_1,block_1>>>(A_dev,B_dev,nx,ny);
        break;
  case 5:

        transformNaiveColUnroll<<<grid_1,block_1>>>(A_dev,B_dev,nx,ny);
        break;
  case 6:
        transformNaiveRowDiagonal<<<grid,block>>>(A_dev,B_dev,nx,ny);
        break;
  case 7:
        transformNaiveColDiagonal<<<grid,block>>>(A_dev,B_dev,nx,ny);
        break;
  default:
    break;
  }
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf(" Time elapsed %f sec\n",iElaps);
  CHECK(hipMemcpy(B_host,B_dev,nBytes,hipMemcpyDeviceToHost));
  checkResult(B_host,B_host,nxy);

  hipFree(A_dev);
  hipFree(B_dev);
  free(A_host);
  free(B_host);
  hipDeviceReset();
  return 0;
}
