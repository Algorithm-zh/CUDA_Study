#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/freshman.h"


void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);
  int power=10;
  if(argc>=2)
    power=atoi(argv[1]);
  int nElem=1<<power;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *res_from_gpu_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_host,*b_host,*res_d;
  double iStart,iElaps;
  dim3 block(1024);
  dim3 grid(nElem/block.x);
  res_from_gpu_h=(float*)malloc(nByte);
  float *a_dev,*b_dev;
  //flags的选值为：
  /*
   * cudaHostAllocDefalt 选这个时相当于hipHostMalloc函数,即创建固定内存
   * hipHostMallocPortable 返回能被所有CUDA上下文使用的固定内存
   * hipHostMallocWriteCombined  返回写结合内存，在某些设备上这种内存传输效率更高
   * hipHostMallocMapped  产生零拷贝内存
   */
  CHECK(hipHostAlloc((float**)&a_host,nByte,hipHostMallocMapped));
  CHECK(hipHostAlloc((float**)&b_host,nByte,hipHostMallocMapped));
  CHECK(hipMalloc((float**)&res_d,nByte));
  initialData(a_host,nElem);
  initialData(b_host,nElem);

 //=============================================================//
  iStart = cpuSecond();
  /*
   * 零拷贝内存虽然不需要显式的传递到设备上，但是设备还不能通过pHost直接访问对应的内存地址，
   * 设备需要访问主机上的零拷贝内存，需要先获得另一个地址，这个地址帮助设备访问到主机对应的内存 
   * pDevice就是设备上访问主机零拷贝内存的指针
   */
  CHECK(hipHostGetDevicePointer((void**)&a_dev,(void*) a_host,0));
  CHECK(hipHostGetDevicePointer((void**)&b_dev,(void*) b_host,0));
  sumArraysGPU<<<grid,block>>>(a_dev,b_dev,res_d);
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  iElaps = cpuSecond() - iStart;
 //=============================================================//
  printf("zero copy memory elapsed %lf ms \n", iElaps);
  printf("Execution configuration<<<%d,%d>>>\n",grid.x,block.x);
//-----------------------normal memory---------------------------
  float *a_h_n=(float*)malloc(nByte);
  float *b_h_n=(float*)malloc(nByte);
  float *res_h_n=(float*)malloc(nByte);
  float *res_from_gpu_h_n=(float*)malloc(nByte);
  memset(res_h_n,0,nByte);
  memset(res_from_gpu_h_n,0,nByte);

  float *a_d_n,*b_d_n,*res_d_n;
  CHECK(hipMalloc((float**)&a_d_n,nByte));
  CHECK(hipMalloc((float**)&b_d_n,nByte));
  CHECK(hipMalloc((float**)&res_d_n,nByte));

  initialData(a_h_n,nElem);
  initialData(b_h_n,nElem);
//=============================================================//
  iStart = cpuSecond();
  CHECK(hipMemcpy(a_d_n,a_h_n,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d_n,b_h_n,nByte,hipMemcpyHostToDevice));
  sumArraysGPU<<<grid,block>>>(a_d_n,b_d_n,res_d_n);
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  iElaps = cpuSecond() - iStart;
//=============================================================//
  printf("device memory elapsed %lf ms \n", iElaps);
  printf("Execution configuration<<<%d,%d>>>\n",grid.x,block.x);
//--------------------------------------------------------------------

  /*
   *
      Vector size:1024
      zero copy memory elapsed 0.000339 ms 
      Execution configuration<<<1,1024>>>
      device memory elapsed 0.000034 ms 
      Execution configuration<<<1,1024>>>
      Check result success!

      Vector size:1024
      zero copy memory elapsed 0.000371 ms 
      Execution configuration<<<1,1024>>>
      device memory elapsed 0.000054 ms 
      Execution configuration<<<1,1024>>>
      Check result success!

      Vector size:1024
      zero copy memory elapsed 0.000346 ms 
      Execution configuration<<<1,1024>>>
      device memory elapsed 0.000104 ms 
      Execution configuration<<<1,1024>>>
      Check result success!
      总结来看就是零拷贝内存比设备主存储器更慢
      但是在一些gpu和cpu集成设备上，他们的物理内存是公用的，零拷贝内存会很有效,而离散设备，通过pcie连接的，就很慢
   */
  sumArrays(a_host,b_host,res_h,nElem);
  checkResult(res_h,res_from_gpu_h,nElem);

  hipHostFree(a_host);
  hipHostFree(b_host);
  hipFree(res_d);
  free(res_h);
  free(res_from_gpu_h);

  hipFree(a_d_n);
  hipFree(b_d_n);
  hipFree(res_d_n);

  free(a_h_n);
  free(b_h_n);
  free(res_h_n);
  free(res_from_gpu_h_n);
  return 0;
}
